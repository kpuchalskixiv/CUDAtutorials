#include "hip/hip_runtime.h"
//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>


//
// kernel routine
// 

// brak hipDeviceSynchronize skutkuje ryzykiem odczytania przez cpu danych wciąż jeszcze nie zsynchronizowanych z gpu. 
// Tak jakbyśmy wywołali print(h_x[n]) przed  
//hipMemcpy(h_x,d_x,nsize*sizeof(float),
  //              hipMemcpyDeviceToHost)
// 

__device__ __managed__ float x[16+1];
__global__ void my_first_kernel()
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  x[tid] = (float) threadIdx.x+blockIdx.x;
}
//
// main code
//

int main(int argc, const char **argv)
{
 // float *x;
  int   nblocks, nthreads, nsize, n; 

  // initialise card

  findCudaDevice(argc, argv);

  // set number of blocks, and threads per block

  nblocks  = 2;
  nthreads = 8;
  nsize    = nblocks*nthreads ;

  // allocate memory for array

  //checkCudaErrors(hipMallocManaged(&x, nsize*sizeof(float)));

  // execute kernel
  
  my_first_kernel<<<nblocks,nthreads>>>();
  getLastCudaError("my_first_kernel execution failed\n");

  // synchronize to wait for kernel to finish, and data copied back
  
  hipDeviceSynchronize();

  for (n=0; n<nsize; n++) printf(" n,  x  =  %d  %f \n",n,x[n]);

  // free memory 

  //checkCudaErrors(hipFree(x));

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

  return 0;
}
